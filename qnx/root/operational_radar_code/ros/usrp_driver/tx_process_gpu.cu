//tx_process_gpu.cu
/* This function mixes and upsamples an arbitrary number
of frequency/beam channels onto each of the 16 antenna
channels.  Input rate = ~ 200 kHz, output rate = ~10 MHz.
Upsampling is done via 1st order linear interpolation
Current known limitations: 
    --Only simple beamforming, (i.e. equal
        amplitude and linear phase shift. No sidelobe suppression,etc.)  
    --Same pulse sequence for each freq/beam channel
*/

#include <hip/hip_runtime.h>
#include <complex>
#include <math.h>
#include <stdio.h>
#include <complex.h>

#define TEST 0
#define NRF_SAMPS 1000
//#define NRF_SAMPS 4000
#define NBB_SAMPS 100
#define NANTS 2
//extern int verbose;

/***********************************************************************
 * interpolate_and_multiply() function
 * A function to be used to upsample/interpolate between two bb samples.
 * This vector is then modulated by NCO(s) to bring it to the desired
 * center frequency.  A phase shift is applied to each carrier for 
 * beam-forming
 **********************************************************************/
__global__ void interpolate_and_multiply(
    float* indata, 
    int16_t** outdata, 
    float* radfreqs, 
    float* phase_delays
){
    /*Declare shared memory array for samples.
    Vectors are written into this array, one for each
    frequency/beam channel.  Then the thread block linearly combines
    the frequency/beam channels into a single vector to be
    transmitted on the antenna*/
    __shared__ float irf_samples[2000];
    __shared__ float qrf_samples[2000];

    //Calculate the increment between two adjacent rf samples
    float inc_i;
    float inc_q;
    inc_i = (indata[2*blockIdx.x+2] - indata[2*blockIdx.x]) / blockDim.x;
    inc_q = (indata[2*blockIdx.x+3] - indata[2*blockIdx.x+1]) / blockDim.x;

    /*Calculate the sample's phase value due to NCO mixing and beamforming*/
    float phase = fmod((double)(blockDim.x*blockIdx.x + threadIdx.x)*radfreqs[threadIdx.y], 2*M_PI) + 
        blockIdx.y*phase_delays[threadIdx.y];

    /*Calculate the output sample vectors, one for each freq/beam channel*/
    unsigned int localInx = threadIdx.y*blockDim.x+threadIdx.x;
    irf_samples[localInx] = 
        (indata[2*blockIdx.x] + threadIdx.x*inc_i) * cos(phase) - 
        (indata[2*blockIdx.x+1] + threadIdx.x*inc_q) * sin(phase);
    qrf_samples[localInx] =
        (indata[2*blockIdx.x] + threadIdx.x*inc_i) * sin(phase) +
        (indata[2*blockIdx.x+1] + threadIdx.x*inc_q) * cos(phase);

    /* Now linearly combine all freq/beam channels into a single vector*/
    __syncthreads();
    unsigned int outInx = blockDim.x*blockIdx.x+threadIdx.x;
    if(threadIdx.y == 0){
        for (unsigned int i=1; i<blockDim.y; i++){
            irf_samples[threadIdx.x] += irf_samples[threadIdx.x + i*blockDim.x];
            qrf_samples[threadIdx.x] += qrf_samples[threadIdx.x + i*blockDim.x];
        }
        outdata[blockIdx.y][2*outInx] = (int16_t) (0.95*32768*irf_samples[threadIdx.x]) & 0xfffe;
        outdata[blockIdx.y][2*outInx+1] = (int16_t) (0.95*32768*qrf_samples[threadIdx.x]) & 0xfffe;
    }
    
}

/***********************************************************************
 * tx_process_gpu() function
 * A function to be used to take a single vector of floats (based on 
 * the master tsg sequence), mix and upsample to the required
 * frequencies, and write to the antenna channel vectors pointed 
 * to by outputs_rf.
 **********************************************************************/
void tx_process_gpu(
    float* input_bb,//input vector (same tx sequence for all frequencies, antennas)
    int16_t** outputs_rf, //output vectors [NANTS][NRF_SAMPLES]
    size_t nbb_samples, //number of input samples
    size_t nrf_samples, //number of samples from host cpu to usrp
    float usrp_center_freq,
    float usrp_samp_rate,
    float *center_freqs, //list of center frequencies to mix up (down) to
    float *time_delays,//per-antenna time offset in ns for each beam
    size_t nchannels, // number of beam directions and/or center frequency channels
    size_t nants // number of antennas
){
   printf("Entering tx_process_gpu\n");
   float mixer_freqs[nchannels];
   float phase_delays[nchannels];

   //Calculate mixer frequencies
   printf("calculating mixer freqs\n");
   for(size_t c=0; c<nchannels; c++){
    mixer_freqs[c] = 2*M_PI*(center_freqs[c] - usrp_center_freq) / usrp_samp_rate;
   }

   //Calculate phase delays
   for(size_t c=0; c<nchannels; c++){
    float rad_phase = fmod(2*M_PI*1e-9*time_delays[c]*center_freqs[c], 2*M_PI);
    printf("rad_phase: %f\n", rad_phase);
    phase_delays[c] = rad_phase;
   }

   // Allocate memory for input vectors and copy data to GPU
   printf("allocating input samples\n");
   float* bbvec_d; 
   hipMalloc((void**)&bbvec_d, 2*(nbb_samples+1)*sizeof(float));
   hipMemset(bbvec_d, 0, 2*(nbb_samples+1)*sizeof(float));
   hipMemcpy(bbvec_d, input_bb, 2*nbb_samples*sizeof(float), hipMemcpyHostToDevice);

   // Allocate memory for output vectors and their pointers
   printf("allocating output sample memory\n");
   int16_t** rfvecptrs_d;
   int16_t* rfvecs_d[nants];
   int16_t* rfvecs_h[nants];
   hipMalloc((void***)&rfvecptrs_d, nants*sizeof(int16_t*));
   for (size_t iant=0; iant<nants; iant++){
    hipMalloc((void**)&rfvecs_d[iant], 2*nrf_samples*sizeof(int16_t));
    rfvecs_h[iant] = rfvecs_d[iant];
   }
   hipMemcpy(rfvecptrs_d, rfvecs_h, nants*sizeof(int16_t*), hipMemcpyHostToDevice);

   // Allocate memory for mixer frequencies and copy data to GPU
   float* mxrs_d; 
   hipMalloc((void**)&mxrs_d, nchannels*sizeof(float));
   hipMemcpy(mxrs_d, mixer_freqs, nchannels*sizeof(float), hipMemcpyHostToDevice);

   // Allocate memory for phase delays (beam-forming) and copy data to GPU
   float* pds_d; 
   hipMalloc((void**)&pds_d, nchannels*sizeof(float));
   hipMemcpy(pds_d, phase_delays, nchannels*sizeof(float), hipMemcpyHostToDevice);

   //Launch Kernel
   printf("Launching kernel\n");
   dim3 dimGrid(nbb_samples, nants, 1);
   dim3 dimBlock(nrf_samples/nbb_samples, nchannels, 1);
   interpolate_and_multiply<<<dimGrid,dimBlock>>>(bbvec_d, rfvecptrs_d, mxrs_d, pds_d);

   printf("copy back\n");
   //Copy output data back to host and free memory
   for (int i=0; i<nants; i++){
    hipMemcpy(outputs_rf[i], rfvecs_d[i], 2*nrf_samples*sizeof(int16_t), hipMemcpyDeviceToHost);
    hipFree(rfvecs_d[i]);
   }
   printf("free\n");

   //Free other device memory
   hipFree(rfvecptrs_d);
   hipFree(pds_d);
   hipFree(mxrs_d);
   hipFree(bbvec_d);

   //printf("output samples head:\n");
   ////for (int i=0; i<nrf_samples; i+=1000){
   //int i = 0;
   //int count = 0;
   //while (i<nrf_samples && count < 100){
   // //if (input_bb[2*i] >= 0.1){
   // if (outputs_rf[0][2*i+1] >= 0.1){
   //     count++;
   //     for (int a=0; a<nants; a++){
   //         float mag = cabsf(outputs_rf[a][2*i] + outputs_rf[a][2*i+1]*I);
   //         float phi = cargf(outputs_rf[a][2*i] + outputs_rf[a][2*i+1]*I);
   //         //printf("output %i %i: (%f, %f)\t", a,i,outputs_rf[a][2*i], outputs_rf[a][2*i+1]);
   //         printf("output %i %i: (%f, %f)\t", a,i,mag, phi);
   //     }
   //     //if (input_bb[2*i] >= 0.1){
   //     //    printf("output %i %i: (%f, %f)\n", a,i,input_bb[2*i], input_bb[2*i+1]);
   //     //}
   // printf("\n");
   // }
   // i++;
   //}


}

#if TEST==1
int main(){
    float* input_vec;
    input_vec = (float*) malloc(2*NBB_SAMPS*sizeof(float));
    for (int i=0; i<NBB_SAMPS; i++){
        //input_vec[2*i] = 0;
        //input_vec[2*i+1] = 0;
        //if ((i%10) < 5){
        input_vec[2*i] = (float)i / (NBB_SAMPS);
        input_vec[2*i+1] = 0;
        //}
    }
    int16_t* output_vec[NANTS];
    for (int i=0; i<NANTS; i++){
        output_vec[i] = (int16_t*) malloc(2*NRF_SAMPS*sizeof(int16_t));
    }
    float center_freq[2] = {10.0, 10.2};
    //float center_freq[1] = {10.0};
    float time_delay[2] = {0.123, 0.123};
    //float time_delay = 0.0;

tx_process_gpu(
    input_vec,//input vector (same tx sequence for all frequencies, antennas)
    &output_vec[0], //output vectors [NANTS][NRF_SAMPLES]
    NBB_SAMPS, //rate of input samples
    NRF_SAMPS, //rate of samples from host cpu to usrp
    10,
    10,
    &center_freq[0], //list of center frequencies to mix up (down) to
    &time_delay[0],//per-antenna time offset in ns for each beam
    2, // number of beam directions and/or center frequencies
    NANTS); // number of antennas

    //for (int i=0; i<NRF_SAMPS; i++){
    //    for (int j=0; j<NANTS; j++){
    //        float mag = cabsf(output_vec[j][2*i] + output_vec[j][2*i+1]*I);
    //        float phi = cargf(output_vec[j][2*i] + output_vec[j][2*i+1]*I);
    //        printf("output %i %i: (%f, %f)\t",j, i, mag, phi);
    //    }
    //    printf("\n");
    //}

    return 0;
}
#endif
