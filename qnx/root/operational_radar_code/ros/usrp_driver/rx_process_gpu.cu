#include "hip/hip_runtime.h"
// rx_process_gpu.cu

/*This function takes ~20 antenna channels that are each sampled at a very high rate
(~10 MHz) and mixes, filters, and decimates an arbitrary number of frequency channels
out of each antenna channel.  The input data is 2-dimensional [NANTS][NRFSAMPLES] and
the output data is 3-dimensional [NFREQS][NANTS][NBBSAMPLES].  This operation lends
itself to parallelization, so it is performed on an Nvidia GPU.*/

/*Description of the general strategy can be found at the UAF SuperDARN google drive:
/UAFSuperDARN/General/Notes/SuperDARN_gpu_processing*/

#include <math.h>
#include <stdio.h>
#include <sys/types.h>

#define TEST 0
#define NRFSAMPS 5000
#define NANTS 1
#define NFREQS 1
#define DECIM_RATE 200
#define MAX_BLOCK_SIZE 512

#if TEST == 0
    extern int verbose;
#endif
#if TEST == 1
    int verbose;
#endif

__global__ void multiply_and_add(float ***samples, float ***odata, float **filter)
{
    __shared__ float itemp[1024];//Max number of threads in a block
    __shared__ float qtemp[1024]; 

    unsigned int tid = threadIdx.y*blockDim.x+threadIdx.x;
    unsigned int tfilt;
    unsigned int tsamp;

    //Each block writes exactly a baseband sample for each frequency
    /*blockIdx.x corresponds to time-domain tiling; NBBSAMPS == gridDim.x
    blockIdx.y corresponds to antenna channel tiling; NANTS == gridDim.y*/
    unsigned int output_idx = 2*blockIdx.x;

    // calculate index of sample from global memory samples array
    tsamp = (blockIdx.x*blockDim.x)/4 + 4*threadIdx.x;

    double i0 = (double) samples[threadIdx.y][blockIdx.y][tsamp];
    double q0 = (double) samples[threadIdx.y][blockIdx.y][tsamp+1];
    double i1 = (double) samples[threadIdx.y][blockIdx.y][tsamp+2];
    double q1 = (double) samples[threadIdx.y][blockIdx.y][tsamp+3];

    // get filter values from global memory
    tfilt = 4*threadIdx.x;
    float p0re = filter[threadIdx.y][tfilt];
    float p0im = filter[threadIdx.y][tfilt+1];
    float p1re = filter[threadIdx.y][tfilt+2];
    float p1im = filter[threadIdx.y][tfilt+3];
        
    // mix samples with nco, perform first reduction
    itemp[tid] = p0re * i0 - p0im * q0 + p1re * i1 - p1im * q1;
    qtemp[tid] = p0re * q0 + p0im * i0 + p1re * q1 + p1im * i1;
        
     __syncthreads();

     // parallel reduce samples (could unroll loop for speedup?)
     // Do this as long as the reduction is a power of 2
     unsigned int s, rem;
     s = blockDim.x / blockDim.y;
     rem = blockDim.x % 2;
     while(s > 0 && rem == 0){
        s /= 2;
        if (threadIdx.x < s) {
            itemp[tid] += itemp[tid + s];
            qtemp[tid] += qtemp[tid + s];
        }
        rem = s % 2;

        __syncthreads();
     }
     //// Now do a serial reduction for the remaining
     if(threadIdx.x == 0){
        for(unsigned int i=1; i<s; i++){
           itemp[tid] += itemp[tid + i];
           qtemp[tid] += qtemp[tid + i];
        }
     }
     __syncthreads();

     if (threadIdx.x == 0) {
        odata[threadIdx.y][blockIdx.y][output_idx] = (float) itemp[tid];
        odata[threadIdx.y][blockIdx.y][output_idx+1] = (float) qtemp[tid];
     }
}     


__global__ void multiply_mix_add(int16_t **samples, float ***odata, float **filter)
{
    __shared__ float itemp[1024];
    __shared__ float qtemp[1024]; 

    unsigned int tid = threadIdx.y*blockDim.x+threadIdx.x;
    unsigned int tfilt;
    unsigned int tsamp;

    //Each block writes exactly a baseband sample for each frequency
    /*blockIdx.x corresponds to time-domain tiling; NBBSAMPS == gridDim.x
    blockIdx.y corresponds to antenna channel tiling; NANTS == gridDim.y
    threadIdx.y corresponds to frequency channel tiling; NFREQS == blockDim.y*/
    unsigned int output_idx = 2*blockIdx.x;

    // calculate index of sample from global memory samples array
    tsamp = (blockIdx.x*blockDim.x) + 4*threadIdx.x;
    //double i0 = (double) samples[blockIdx.y][tsamp];
    //double q0 = (double) samples[blockIdx.y][tsamp+1];
    //double i1 = (double) samples[blockIdx.y][tsamp+2];
    //double q1 = (double) samples[blockIdx.y][tsamp+3];

    // get filter values from global memory
    tfilt = 4*threadIdx.x;
    //double p0re = filter[threadIdx.y][tfilt];
    //double p0im = filter[threadIdx.y][tfilt+1];
    //double p1re = filter[threadIdx.y][tfilt+2];
    //double p1im = filter[threadIdx.y][tfilt+3];
        
    // mix samples with nco, perform first reduction
    //itemp[tid] = p0re * i0 - p0im * q0 + p1re * i1 - p1im * q1;
    itemp[tid] = 
        filter[threadIdx.y][tfilt] * samples[blockIdx.y][tsamp] - 
        filter[threadIdx.y][tfilt+1] * samples[blockIdx.y][tsamp+1] +
        filter[threadIdx.y][tfilt+2] * samples[blockIdx.y][tsamp+2] -
        filter[threadIdx.y][tfilt+3] * samples[blockIdx.y][tsamp+3];
    //qtemp[tid] = p0re * q0 + p0im * i0 + p1re * q1 + p1im * i1;
    qtemp[tid] = 
        filter[threadIdx.y][tfilt] * samples[blockIdx.y][tsamp+1] + 
        filter[threadIdx.y][tfilt+1] * samples[blockIdx.y][tsamp] +
        filter[threadIdx.y][tfilt+2] * samples[blockIdx.y][tsamp+3] +
        filter[threadIdx.y][tfilt+3] * samples[blockIdx.y][tsamp+2];
        
     __syncthreads();

     /* Example: dmrate==100,
     100 -> 50 -> 25 -> 5 -> 1*/
     // parallel reduce samples (could unroll loop for speedup?)
     // Do this as long as the reduction is a power of 2
     unsigned int s, rem;
     s = blockDim.x;
     rem = blockDim.x % 2;
     while(s > 0 && rem == 0){
        s /= 2;
        if (threadIdx.x < s) {
            itemp[tid] += itemp[tid + s];
            qtemp[tid] += qtemp[tid + s];
        }
        rem = s % 2;

        __syncthreads();
     }

     // Do this as long as the reduction is a power of 5
     rem = s % 5;
     while(s > 0 && rem == 0){
        s /= 5;
        if (threadIdx.x < s) {
            itemp[tid] = itemp[tid] + itemp[tid + s] + itemp[tid+2*s] + itemp[tid+3*s] + itemp[tid+4*s];
            qtemp[tid] = qtemp[tid] + qtemp[tid + s] + qtemp[tid+2*s] + qtemp[tid+3*s] + qtemp[tid+4*s];
        }
        rem = s % 5;

        __syncthreads();
     }

     // Now do a serial reduction for the remaining
     if(threadIdx.x == 0){
        for(unsigned int i=1; i<s; i++){
           itemp[tid] += itemp[tid + i];
           qtemp[tid] += qtemp[tid + i];
        }
     }
     __syncthreads();

     if (threadIdx.x == 0) {
        /*Now do phase adjustment on the output samples*/
        // phase increment of the NCO can be calculated from two adjacent filter taps
        //double phase_inc = atan(p1im / p1re) - atan(p0im / p0re);
        double phase_inc = 
            atan(filter[threadIdx.y][tfilt+3] / filter[threadIdx.y][tfilt+2]) - 
            atan(filter[threadIdx.y][tfilt+1] / filter[threadIdx.y][tfilt]);

        /*Phase remainder exists because the NCO oscillator 
        may not complete an exact 360% rotation in a filter window*/
        double phi_rem = blockIdx.x*fmod((0.5*blockDim.x) * phase_inc, 2*M_PI);

        double ltemp = (double) itemp[tid];
        itemp[tid] = itemp[tid] * cos(phi_rem) - qtemp[tid] * sin(phi_rem);
        qtemp[tid] = ltemp * sin(phi_rem) + qtemp[tid] * cos(phi_rem);

        //deciding the output
        odata[threadIdx.y][blockIdx.y][output_idx] = (float) itemp[tid];
        //odata[threadIdx.y][blockIdx.y][output_idx] = threadIdx.x;
        odata[threadIdx.y][blockIdx.y][output_idx+1] = (float) qtemp[tid];
        //odata[threadIdx.y][blockIdx.y][output_idx+1] = 20;
     }
}     

void rx_process_gpu(
    int16_t **rx_buff_ptrs,
    float ***client_buff_ptr,
    size_t nrf_samples, 
    size_t nbb_samples, //perhaps this should be different for each center frequency..
    size_t nfreqs, 
    size_t nants, 
    float rf_sample_rate,
    float client_sample_rate,
    float* rx_freqs // rx center frequencies
    //std::vector<float> center_freqs, // center frequencies
    //std::vector<float> bws // bandwidth of each center frequency
){
    int trblshoot = 0;
    struct timespec t0, t1;
    struct timespec tick, tock;
    float elapsed_t, elapsed_proc_t;
    int dmrate = nrf_samples / nbb_samples;
    int dmrate0 = dmrate;
    //nants *= 10;
    if (trblshoot) printf("Entered rx_process_gpu()\n");


    //if (dmrate*nfreqs > MAX_BLOCK_SIZE){
    //    dmrate0 = MAX_BLOCK_SIZE / nfreqs;
    //    dmrate1 = dmrate / dmrate0;
    //    extra_stages=1;
    //}
    dmrate0 = 100;
    int dmrate1 = 2*dmrate / dmrate0;
        
    int ntaps0 = 2*dmrate0; //The coarse filter length is 2x the decimation rate
    int ntaps1 = 8*dmrate1; //The fine filter length is 8x the decimation rate

    float filter_taps0[nfreqs][ntaps0][2];
    float filter_taps1[nfreqs][ntaps1][2];
    double NCO0[nfreqs][ntaps0][2]; //filter taps sent to the GPU

    /*Calculate the per-sample phase shift of each NCO*/
    double radfreq0[nfreqs];
    for (size_t i=0;i<nfreqs;i++){
	    radfreq0[i] = double (2*M_PI* rx_freqs[i]) / rf_sample_rate;
    }

    /*Rectangular-window for coarse filtering.  
    [TODO: design Kaiser (beta=5) or similar window for better performance.
    See Google drive doc SuperDARN_GPU_processing]*/
    for (int i=0;i<ntaps0;i++){
            filter_taps0[0][i][0] = 1./ntaps0; //Q-component is zero
            //filter_taps0[0][i][0] = (0.54-0.46*cos((2*M_PI*((float)(i)+0.5))/ntaps0));
            
            filter_taps0[0][i][1] = 0; //Q-component is zero
    }

    /*Mix each filter up(down) to the desired pass-band frequency.
    Use the first set of coefficients as the reference*/
    double ftemp;
    for (size_t i=1;i<nfreqs;i++){
    	for (int j=0;j<ntaps0;j++){
                NCO0[i][j][0] = cos(-j*radfreq0[i]);
                NCO0[i][j][1] = sin(-j*radfreq0[i]);
                ftemp = filter_taps0[0][j][0];
    	        filter_taps0[i][j][0] = NCO0[i][j][0] * filter_taps0[0][j][0] - NCO0[i][j][1] * filter_taps0[0][j][1];
    	        filter_taps0[i][j][1] = NCO0[i][j][0] * filter_taps0[0][j][1] + NCO0[i][j][1] * ftemp;
	    }
    }
    /*Now go back to the first set of coefficients and modulate that*/
    for (int j=0;j<ntaps0;j++){
            NCO0[0][j][0] = cos(-j*radfreq0[0]);
            NCO0[0][j][1] = sin(-j*radfreq0[0]);
            ftemp = filter_taps0[0][j][0];
            filter_taps0[0][j][0] = NCO0[0][j][0] * filter_taps0[0][j][0] - NCO0[0][j][1] * filter_taps0[0][j][1];
            filter_taps0[0][j][1] = NCO0[0][j][0] * filter_taps0[0][j][1] + NCO0[0][j][1] * ftemp;
	}



    if (trblshoot) printf("Allocating and copying filter tap data\n");
    /*Allocate device memory for filter taps and copy data into it*/
    float *taps_dptr[nfreqs];// = (float*) malloc(nfreqs*sizeof(float*));
    float* taps_ptrs[nfreqs];
    float **taps_ptr_dptr;
    hipMalloc( (void***)&taps_ptr_dptr, nfreqs*sizeof(float*)); //for the pointers to the taps
    for (int ifreq=0;ifreq<nfreqs;ifreq++){
        hipMalloc( (void**)&taps_dptr[ifreq], ntaps0*sizeof(float)*2); //for the taps themselves
        hipMemcpy(
             taps_dptr[ifreq], 
             &filter_taps0[ifreq][0][0],
             ntaps0*2*sizeof(float),
             hipMemcpyHostToDevice);
        taps_ptrs[ifreq] = taps_dptr[ifreq];
    }
    hipMemcpy(
         taps_ptr_dptr,
         taps_ptrs,
         nfreqs*sizeof(float*),
         hipMemcpyHostToDevice);

    if (trblshoot) printf("Allocating and copying input data\n");
    /*Allocate device memory for input data and copy data into it*/
    /* TODO: this should be a function in recv_and_hold(); i.e. data
    should be ready to rock by the time this function is called*/
    int16_t *indata_vp_d[nants];// = malloc(nfreqs*sizeof(int16_t*));
    int16_t *indata_p2vp_h[nants];// = (int16_t**) malloc(nants*sizeof(int16_t*));// = (int16_t**) malloc(nfreqs*sizeof(int16_t*));
    int16_t** indata_p2vp_d;
    hipMalloc( (void***)&indata_p2vp_d, nants*sizeof(int16_t*));
    for (int iant=0; iant<nants; iant++){
        hipMalloc( (void**)&indata_vp_d[iant], (nrf_samples+ntaps0)*sizeof(int16_t)*2);
        hipMemset(indata_vp_d[iant], 0, (nrf_samples+ntaps0)*sizeof(int16_t)*2);
        //hipMemset(indata_vp_d[iant], 0, ntaps0*sizeof(int16_t));
        //hipMemset(indata_vp_d[iant]+2*nrf_samples+ntaps0, 0, ntaps0*sizeof(int16_t));
        //indata_vp_d[iant] += dmrate0;
        if (trblshoot) printf("rf data to GPU. nrfsamples: %i\n", nrf_samples);
        hipMemcpy(
                indata_vp_d[iant] + ntaps0,
                rx_buff_ptrs[iant], 
                nrf_samples*2*sizeof(int16_t),
                hipMemcpyHostToDevice);
        indata_p2vp_h[iant] = indata_vp_d[iant] + dmrate0;
    }
    if (trblshoot) printf("rf data ptrs to GPU\n");
    hipMemcpy(
            indata_p2vp_d,
            indata_p2vp_h,
            nants*sizeof(int16_t*),
            hipMemcpyHostToDevice);
        
    if (trblshoot) printf("Allocating output data\n");
    /*Allocate device memory for output data*/
    float* outdata_vp_d[nfreqs][nants];// = malloc(nfreqs*sizeof(float*));
    float* outdata_vp_h[nfreqs][nants];// = (float**) malloc(nants*sizeof(float*));// = (float**) malloc(nfreqs*sizeof(float*));
    float** outdata_p2vp_d[nfreqs];
    float** outdata_p2vp_h[nfreqs];// = (float**) malloc(nants*sizeof(float*));// = (float**) malloc(nfreqs*sizeof(float*));
    float*** outdata_p2channels_d;
    hipMalloc( (void****)&outdata_p2channels_d, nfreqs*sizeof(float**));
    for (int ifreq=0; ifreq<nfreqs; ifreq++){
        hipMalloc( (void***)&outdata_p2vp_d[ifreq], nants*sizeof(float*));
        //outdata_p2vp_h[ifreq] = outdata_p2vp_d[ifreq];
        for (int iant=0; iant<nants; iant++){
            hipMalloc( (void**)&outdata_vp_d[ifreq][iant], (nrf_samples/dmrate0)*sizeof(float)*2);
            outdata_vp_h[ifreq][iant] = outdata_vp_d[ifreq][iant];
        }
        hipMemcpy(
                outdata_p2vp_d[ifreq],
                &outdata_vp_h[ifreq][0],
                nants*sizeof(float*),
                hipMemcpyHostToDevice);
    outdata_p2vp_h[ifreq] = outdata_p2vp_d[ifreq];
    }
    hipMemcpy(
            outdata_p2channels_d,
            outdata_p2vp_h,
            nfreqs*sizeof(float**),
            hipMemcpyHostToDevice);

    //hipMalloc( (void**)&outdata_dptr, nfreqs*nbb_samples*nants*sizeof(float)*2);
    //float output_buffer[NFREQS][NANTS][nbb_samples][2];// = (float*) malloc(nbb_samples*nants*nfreqs*2*sizeof(float));


    dim3 dimGrid(nrf_samples/dmrate0+1,nants,1);
    dim3 dimBlock(ntaps0/2,nfreqs,1);

    if (trblshoot){
        printf("nrfsamples: %i\n", nrf_samples);
        printf("decimation rate 0: %i\n", dmrate0);
        printf("nimtsamples: %i\n", nrf_samples/dmrate0);
        printf("decimation rate 1: %i\n", dmrate1);
        printf("nbbsamples: %i\n", nbb_samples);
        printf("dimGrid: (%i, %i, %i)\n", nrf_samples/dmrate0, nants, 1);
        printf("dimBlock: (%i, %i, %i)\n", ntaps0/2, nfreqs, 1);
    }


    //printf("About to print output of first stage\n");
    //for (int i=0; i<nrf_samples/dmrate0; i+=100){
    //    //printf("rx_process_gpu: output samples head\n");
    //    int iant =0, ifreq=0;
    //    printf("output %i, %i: (%i, %i)\n",ifreq, iant,
    //    (int) outdata_p2vp_h[0][0][2*i],
    //    (int) outdata_p2vp_h[0][0][2*i+1]);
    //  printf("\n\n");
    //}

    clock_gettime(CLOCK_MONOTONIC, &tick);
    multiply_mix_add<<<dimGrid, dimBlock>>>(indata_p2vp_d, outdata_p2channels_d, taps_ptr_dptr);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &tock);



    /* Uncomment the following to take a sneak-peak at the intermediate stage data*/

    
    if (trblshoot>1 | TEST==1){
        float* vps[nfreqs][nants];
        for (int ifreq=0; ifreq<nfreqs; ifreq++){
            for (int iant=0; iant<nants; iant++){
                vps[ifreq][iant] = (float*) malloc(nrf_samples/dmrate0 * 2*sizeof(float));
                hipMemcpy( 
                    vps[ifreq][iant], 
                    outdata_vp_d[ifreq][iant], 
                    (nrf_samples/dmrate0) * 2*sizeof(float),
                    hipMemcpyDeviceToHost);
            }
        }
        printf("\nPrinting intermediate stage data\n\n");
        for (int ifreq=0; ifreq<nfreqs; ifreq++){
            for (int iant=0; iant<nants; iant++){
                if (iant ==0){
                    for (int isamp=0; isamp < nrf_samples/dmrate0; isamp++){
                        printf("%i, %i, (%.1f, %.1f)\n", ifreq, iant, vps[ifreq][iant][2*isamp], vps[ifreq][iant][2*isamp+1]);
                    }
                }
            }
        }
    }
    
    
    

    /*

    First Stage Done!!  Now do a finer filtering at the much-lower sample rate

    */

    /* Free memory related to raw rf samples*/
    hipFree(indata_p2vp_d);
    for (int iant=0; iant<nants; iant++){
        hipFree(indata_vp_d[iant]);
    }

    ntaps1 = 8*dmrate1;

    /*Allocate device memory for 2nd stage input data,
    copy output of previous stage into new location,
    and free all memory related to 1st filter stage*/ 
    float* imt_vp_d[nfreqs][nants];
    float* imt_vp_h[nfreqs][nants];
    float** imt_p2vp_d[nfreqs];
    float** imt_p2vp_h[nfreqs];
    float*** imt_p2channels_d;
    hipMalloc( (void****)&imt_p2channels_d, nfreqs*sizeof(float**));
    hipFree(outdata_p2channels_d);
    for (int ifreq=0; ifreq<nfreqs; ifreq++){
        hipMalloc( (void***)&imt_p2vp_d[ifreq], nants*sizeof(float*));
        hipFree(outdata_p2vp_d[ifreq]);
        for (int iant=0; iant<nants; iant++){
            hipMalloc( (void**)&imt_vp_d[ifreq][iant], ((nrf_samples/dmrate0)+ntaps1)*sizeof(float)*2);
            hipMemset( imt_vp_d[ifreq][iant], 0x00, ((nrf_samples/dmrate0)+ntaps1)*sizeof(float)*2);
            hipMemcpy( 
                imt_vp_d[ifreq][iant]+ntaps1, 
                outdata_vp_d[ifreq][iant], 
                (nrf_samples/dmrate0) * 2*sizeof(float),
                hipMemcpyDeviceToDevice);
            //imt_vp_d[ifreq][iant] += ntaps1;
            //imt_vp_h[ifreq][iant] = imt_vp_d[ifreq][iant] + ntaps1;
            imt_vp_h[ifreq][iant] = imt_vp_d[ifreq][iant];
            hipFree(outdata_vp_d[ifreq][iant]);
        }
        hipMemcpy(
                imt_p2vp_d[ifreq],
                imt_vp_h[ifreq],
                nants*sizeof(float*),
                hipMemcpyHostToDevice);
    imt_p2vp_h[ifreq] = imt_p2vp_d[ifreq];
    }
    hipMemcpy(
            imt_p2channels_d,
            imt_p2vp_h,
            nfreqs*sizeof(float**),
            hipMemcpyHostToDevice);

    /* Calculate filtertaps for second filter stage
    The filters for each channel are probably the same but in certain applications
    they might not be.  Take care of that here if you like*/
    for (int ifreq=0; ifreq<nfreqs; ifreq++){
        for (int i=0;i<ntaps1;i++){
                double x = 8*(2*M_PI*((float)i/ntaps1) - M_PI);
                filter_taps1[ifreq][i][0] = (0.54-0.46*cos((2*M_PI*((float)(i)+0.5))/ntaps1))
                	*sin(x)/(x);
                //filter_taps1[ifreq][i][0] = 1./ntaps1; //Q-component is zero
                filter_taps1[ifreq][i][1] = 0; //Q-component is zero
                //printf("filter tap %i: %f\n", i, filter_taps1[ifreq][i][0]);
        }
    }
    filter_taps1[0][ntaps1/2][0]=1./ntaps1; //handle the divide-by-zero condition

    /*Allocate device memory for filter taps and copy data into it*/
    hipFree(taps_ptr_dptr);
    for (int ifreq=0;ifreq<nfreqs;ifreq++)
        hipFree(taps_dptr[ifreq]);

    hipMalloc( (void***)&taps_ptr_dptr, nfreqs*sizeof(float*)); //for the pointers to the taps
    for (int ifreq=0;ifreq<nfreqs;ifreq++){
        hipMalloc( (void**)&taps_dptr[ifreq], ntaps1*sizeof(float)*2); //for the taps themselves
        hipMemcpy(
             taps_dptr[ifreq], 
             filter_taps1[ifreq][0],
             ntaps1*2*sizeof(float),
             hipMemcpyHostToDevice);
        taps_ptrs[ifreq] = taps_dptr[ifreq];
    }
    hipMemcpy(
         taps_ptr_dptr,
         taps_ptrs,
         nfreqs*sizeof(float*),
         hipMemcpyHostToDevice);

    /*Allocate device memory for final output data*/
    float* fout_vp_d[nfreqs][nants];// = malloc(nfreqs*sizeof(float*));
    float* fout_vp_h[nfreqs][nants];// = (float**) malloc(nants*sizeof(float*));// = (float**) malloc(nfreqs*sizeof(float*));
    float** fout_p2vp_d[nfreqs];
    float** fout_p2vp_h[nfreqs];// = (float**) malloc(nants*sizeof(float*));// = (float**) malloc(nfreqs*sizeof(float*));
    float*** fout_p2channels_d;
    hipMalloc( (void****)&fout_p2channels_d, nfreqs*sizeof(float**));
    for (int ifreq=0; ifreq<nfreqs; ifreq++){
        hipMalloc( (void***)&fout_p2vp_d[ifreq], nants*sizeof(float*));
        //outdata_p2vp_h[ifreq] = outdata_p2vp_d[ifreq];
        for (int iant=0; iant<nants; iant++){
            hipMalloc( (void**)&fout_vp_d[ifreq][iant], nbb_samples*sizeof(float)*2);
            fout_vp_h[ifreq][iant] = fout_vp_d[ifreq][iant];
        }
        hipMemcpy(
                fout_p2vp_d[ifreq],
                fout_vp_h[ifreq],
                nants*sizeof(float*),
                hipMemcpyHostToDevice);
    fout_p2vp_h[ifreq] = fout_p2vp_d[ifreq];
    }
    hipMemcpy(
            fout_p2channels_d,
            fout_p2vp_h,
            nfreqs*sizeof(float**),
            hipMemcpyHostToDevice);

    if (trblshoot){
        printf("nimtsamples: %i\n", nrf_samples/dmrate0);
        printf("nbbsamples: %i\n", nbb_samples);
        printf("decimation rate 1: %i\n", dmrate1);
        printf("dimGrid: (%i, %i, %i)\n", nbb_samples, nants, 1);
        printf("dimBlock: (%i, %i, %i)\n", ntaps1/2, nfreqs, 1);
    }

    dim3 dimGrid1(nbb_samples,nants,1);
    dim3 dimBlock1(ntaps1/2,nfreqs,1);
    

    clock_gettime(CLOCK_MONOTONIC, &t0);
    multiply_and_add<<<dimGrid1, dimBlock1>>>(imt_p2channels_d, fout_p2channels_d, taps_ptr_dptr);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &t1);

    //here is the output
    for (int ifreq=0; ifreq<nfreqs; ifreq++){
        hipFree(taps_dptr[ifreq]);
        hipFree(fout_p2vp_d[ifreq]);
        hipFree(imt_p2vp_d[ifreq]);
        for (int iant=0; iant<nants; iant++){
        //printf("copying %i samples back to client buffer\n",nbb_samples);
        //hipMemcpy(
        //    client_buff_ptr[ifreq][iant],
        //    imt_vp_d[ifreq][iant], 
        //    nbb_samples*2*sizeof(float),
        //    hipMemcpyDeviceToHost);
        
        hipMemcpy(
            client_buff_ptr[ifreq][iant],
            fout_vp_d[ifreq][iant], 
            nbb_samples*2*sizeof(float),
            hipMemcpyDeviceToHost);

        /* Uncomment the following to get a peak at the output samples*/
        if (trblshoot > 1 | TEST==1){
            for(int i=0;i<nbb_samples;i+=1){
                    if (iant == 0){
                    printf("output %i, %i, %i: (%i, %i)\n",ifreq, iant, i,
                         (int) client_buff_ptr[ifreq][iant][2*i],
                         (int) client_buff_ptr[ifreq][iant][2*i+1]);
                    }
            }
        }

        hipFree(fout_vp_d[ifreq][iant]);
        hipFree(imt_vp_d[ifreq][iant]);
        //printf("rx_process_gpu: output samples head\n");
        }
      //printf("\n\n");
    }
    hipFree(fout_p2channels_d);
    hipFree(imt_p2channels_d);
    hipFree(taps_ptr_dptr);

    elapsed_t = (1e9*t1.tv_sec + t1.tv_nsec) - (1e9*t0.tv_sec + t0.tv_nsec);
    elapsed_proc_t = (1e9*tock.tv_sec + tock.tv_nsec) - (1e9*tick.tv_sec + tick.tv_nsec);
    if(verbose>0) printf("GPU: first stage time: %.2f ms, second stage time: %.2f ms, total time: %.2f ms\n",
                            elapsed_proc_t/1e6, elapsed_t/1e6, (elapsed_proc_t+elapsed_t)/1e6);
    //if(verbose>0) printf("GPU: total time (includes data transfers): %f ms\n", elapsed_t/1e6);


}



#if TEST == 1
int main(){
    int16_t *fake_samples;//[NANTS][NRFSAMPS+DECIM_RATE][2];
    int16_t **vec_ptrs;

    float** ant_channels[NFREQS];
    float* float_vecs[NFREQS][NANTS];
    float*** output_buffer = (float***) malloc(NFREQS*sizeof(float**));
    for (int ifreq=0; ifreq<NFREQS; ifreq++){
        ant_channels[ifreq] = (float**) malloc(NANTS*sizeof(float*));
        for (int iant=0; iant<NANTS; iant++){
            float_vecs[ifreq][iant] = (float*) malloc(NRFSAMPS/DECIM_RATE * 2*sizeof(float));
        }
        ant_channels[ifreq] = float_vecs[ifreq];
    }
    output_buffer = ant_channels;
            

    fake_samples = (int16_t*) malloc(NANTS*(NRFSAMPS)*2*sizeof(int16_t));

    for(int iant=0; iant<NANTS; iant++){
        for (int i=0;i<NRFSAMPS;i++){
            fake_samples[iant*((NRFSAMPS)*2)+2*i] = i;//sin(M_PI*i/4);
            fake_samples[iant*((NRFSAMPS)*2)+2*i+1] = 0;
            printf("fake_samples %i: %i\n", fake_samples[iant*((NRFSAMPS)*2)+2*i]);
        }
    }

    vec_ptrs = (int16_t**) malloc(NANTS*sizeof(int16_t *));
    printf("building vec ptrs\n");

    for (int i=0; i<NANTS; i++)
        *(vec_ptrs+i) = fake_samples + i*((NRFSAMPS)*2);

    printf("built vec ptrs\n");

    struct timespec tick, tock;
    float elapsed;
    float center_freqs[NFREQS];
    for (int i=0;i<NFREQS;i++){
        //center_freqs[i] = M_PI/4;
        center_freqs[i] = 0;
    }
    clock_gettime(CLOCK_MONOTONIC, &tick);
    rx_process_gpu(
        vec_ptrs,
        output_buffer,
        NRFSAMPS,
        NRFSAMPS/DECIM_RATE,
        NFREQS,
        NANTS,
        NRFSAMPS,
        2,
        &center_freqs[0]);
    clock_gettime(CLOCK_MONOTONIC, &tock);
    elapsed = (1e9*tock.tv_sec + tock.tv_nsec) - (1e9*tick.tv_sec + tick.tv_nsec);

    free(fake_samples);

    //here is the output
    //for (int ifreq=0; ifreq<NFREQS; ifreq++){

    //    printf("copying %i samples back to client buffer\n",NRFSAMPS/DECIM_RATE);
    //    printf("rx_process_gpu: output samples head\n");

    //    for(int i=0;i<NRFSAMPS/DECIM_RATE;i+=1){
    //        for (int iant=0; iant<NANTS; iant++){
    //                printf("output %i, %i: (%f, %f)\t",ifreq, iant,
    //                     (float) output_buffer[ifreq][iant][2*i],
    //                     (float) output_buffer[ifreq][iant][2*i+1]);
    //                    //*(client_buff_ptr+(ifreq*nbb_samples*nants*2) + iant*nbb_samples*2 + 2*i),
    //                    //*(client_buff_ptr+(ifreq*nbb_samples*nants*2) + iant*nbb_samples*2 + 2*i + 1));
    //        }
    //        printf("\n");
    //    }
    //  printf("\n\n");
    //}


    //for (int ifreq=0; ifreq<NFREQS; ifreq++){
    //    for(int i=0;i<NRFSAMPS/DECIM_RATE;i++){
    //        for (int j=0; j<NANTS; j++)
    //            printf("output %i,%i: (%f,%f)\t",ifreq,j,
    //                output_buffer[ifreq][j][i][0],
    //                output_buffer[ifreq][j][i][1]);
    //    printf("\n");
    //    }
    //  printf("\n\n");
    //}

    printf("processesd fake samples\n");
    printf("elapsed time: %f ms\n", elapsed/1e6);

    return 0;
}
#endif
